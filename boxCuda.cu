#include "hip/hip_runtime.h"
/**
 * @file box.cpp
 * @brief Implements the Box class methods.
 * 
 * This file provides the implementation of the Box class, including
 * methods for particle management, force calculations, and simulation execution.
 */

#include "boxCuda.cuh"

/**
 * Initializes the box with given dimensions
 * 
 * @param x_length Length of the box in the x-direction.
 * @param y_length Length of the box in the y-direction.
 * @param z_length Length of the box in the z-direction.
 */
Box::Box(const double x_length, const double y_length, const double z_length) 
    : Lx(x_length), Ly(y_length), Lz(z_length), N(0), host_particles(nullptr), gpu_particles(nullptr) {
}

/**
 * Dellocate memory for the host and GPU arrays
 */
Box::~Box() {
    delete[] host_particles;
    hipFree(gpu_particles);
}

/**
 * Adds the given particle to the box.
 * Uses /f$ R^2 = x_{ij}^2 + y_{ij}^2 + z_{ij}^2 /f$ where /f$ a_{ij} = a_i - a_j /f$.
 * If /f$ R^2 < 0.25 /f$, the particle is removed.
 * 
 * @param p The particle to be added to the box.
 * @return 'true' if the particle was added sucessfully, 'false' if not.
 */
bool Box::addParticle(Particle& p) {
    particles.push_back(p);             // append particle to vector

    for (int j = 0; j < N; j++) {       // iterate through every other particle
        Particle& p_j = particles[j];
        double d, R2 = 0.0;
        for (int m = 0; m < 3; m++) {
            d = p.r[m] - p_j.r[m];
            R2 += d*d;                  
        }
        if (R2 < 0.25) {
            particles.pop_back();       // remove particle in last position if R^2 < 0.25
            return false;
        }
    }
    N++;                                // Increment number of particles if added successfully
    return true;
}

/**
 * @brief CUDA Kernel to calculate the force on a particle exerted by all other particles
 * 
 * Uses /f$ \mathbf{F}_i = - \nabla \sum_{\substack{j \neq i}}^{N} \phi_{ij}./f$ to calculate
 * the force on particle i, where \frac{d \phi_{ij}}{dx} = -24 \epsilon x_{ij} \frac{\sigma^6}{r_{ij}^8} \left( \frac{2\sigma^6}{r_{ij}^6} - 1 \right).
 * Using Newton's Law, the force exerted on p_i by p_j is equal but opposite to the force exerted on p_j by p_i.
 * Therefore, the total force is calculated on p_i, but also the force exerted by p_i on other particles.
 * 
 * @param gpu_particles The gpu array to be used
 * @param N The total number of particles
 */
__global__ 
void calculateF_i(double* gpu_particles, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // calculate array index
    if (i < N) {                                    // Ensure thread only executes if it is less than N
        double Fx = 0.0, Fy = 0.0, Fz = 0.0;
        double eps, sig, dx, dy, dz, r_ij2, dphi_dx, sig_rij, inv_rij2;

        for (int j = 0; j < N; j++) {
            if (i != j) {       // skip identical particle index

                if (gpu_particles[10 * i + 9] == gpu_particles[10 * j + 9]) {         // Find the parameters based on the two particles
                    if (gpu_particles[10 * i + 9] == 1.0) {
                        eps = 3.0; 
                        sig = 1.0;
                    } else {
                        eps = 60.0;
                        sig = 3.0;
                    }
                } else {
                    eps = 15.0;
                    sig = 2.0;
                }

                dx = gpu_particles[10 * i] - gpu_particles[10 * j];
                dy = gpu_particles[10 * i + 1] - gpu_particles[10 * j + 1];
                dz = gpu_particles[10 * i + 2] - gpu_particles[10 * j + 2];

                r_ij2 = dx * dx + dy * dy + dz * dz;
                inv_rij2 = 1.0 / r_ij2;                                                             // Calculate 1/r^2
                sig_rij = (sig * sig * sig * sig * sig * sig) * inv_rij2 * inv_rij2 * inv_rij2;     // Calcualte (sigma/r)^6
                dphi_dx = 24.0 * eps * sig_rij * (2 * sig_rij - 1) * inv_rij2;                       // Calculate the constant in the force equation

                Fx += dphi_dx * dx;
                Fy += dphi_dx * dy;
                Fz += dphi_dx * dz;
            }
        }
        gpu_particles[10 * i + 6] = Fx;
        gpu_particles[10 * i + 7] = Fy;
        gpu_particles[10 * i + 8] = Fz;
    }
}

/**
 * @brief CUDA Kernel to update positions
 * 
 * @param gpu_particles gpu array 
 * @param dt time step
 * @param N number of particles
 */
__global__ 
void updatePositions(double *gpu_particles, double dt, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    gpu_particles[10 * i]     += dt * gpu_particles[10 * i + 3];
    gpu_particles[10 * i + 1] += dt * gpu_particles[10 * i + 4];
    gpu_particles[10 * i + 2] += dt * gpu_particles[10 * i + 5];
}

/**
 * @brief CUDA Kernel to update velocities and apply boundary conditions
 * 
 * Uses the equation \f$ v = v + (F/m) \cdot dt \f$.
 * Also applies boundary conditions to prevent particles from exiting the simulation domain.
 * 
 * @param gpu_particles gpu array 
 * @param dt time step
 * @param N number of particles
 * @param Lx Box dimension in the x-direction.
 * @param Ly Box dimension in the y-direction.
 * @param Lz Box dimension in the z-direction.
 */
__global__ 
void updateVelocities(double *gpu_particles, double dt, int N, double Lx, double Ly, double Lz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        double mass = gpu_particles[10 * i + 9];
        double L[3] = {Lx, Ly, Lz};

        for (int dim = 0; dim < 3; dim++) {
            gpu_particles[10 * i + dim + 3] += dt * gpu_particles[10 * i + dim + 6] / mass;  // v = v + dt * F/m
            if (gpu_particles[10 * i + dim] < 0) {
                gpu_particles[10 * i + dim] = -gpu_particles[10 * i + dim]; 
                gpu_particles[10 * i + 3 + dim]  = abs(gpu_particles[10 * i + 3 + dim] );
            }
            if (gpu_particles[10 * i + dim] > L[dim]) {
                gpu_particles[10 * i + dim] = 2 * L[dim] - gpu_particles[10 * i + dim];
                gpu_particles[10 * i + 3 + dim] = -abs(gpu_particles[10 * i + 3 + dim]);
            }
        }
    }
}

__global__ 
void computeKineticEnergy(double* gpu_particles, double* gpu_KE, int N) {
    __shared__ double shared_KE[256];  // Shared memory for parallel reduction
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    double KE = 0.0;
    if (i < N) {
        double vx = gpu_particles[10 * i + 3];
        double vy = gpu_particles[10 * i + 4];
        double vz = gpu_particles[10 * i + 5];
        double mass = gpu_particles[10 * i + 9];

        KE = 0.5 * mass * (vx * vx + vy * vy + vz * vz);
    }
    
    shared_KE[tid] = KE;
    __syncthreads();

    // Parallel reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_KE[tid] += shared_KE[tid + s];
        }
        __syncthreads();
    }

    // Write to global memory
    if (tid == 0) {
        atomicAdd(gpu_KE, shared_KE[0]);
    }
}


__global__ 
void scaleVelocities(double* gpu_particles, double lambda, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        gpu_particles[10 * i + 3] *= lambda;
        gpu_particles[10 * i + 4] *= lambda;
        gpu_particles[10 * i + 5] *= lambda;
    }
}


/**
 * The simulation is run for T seconds where the particles are updated every timesetep.
 * The kinetic energy is written to a text file every 0.1 seconds.
 * The particle positions in the x-y plane are written to a text file for the test cases.
 * 
 * @param dt The timestep to update the particles
 * @param T The total time to run the simulation
 * @param temp The temperature to be fixed to, however, if equal to -1 not kept constant
 * @param ic_random 'true' if the initial condition is ic_random, 'false' if not, used for writing to text files.
 * @param ic The initial condition being run, used to name the text files appropriately as ic.txt.
 */
void Box::runSimulation(double dt, double T, double temp, bool ic_random, string ic) {
    string particle_file = "Particle_Data/Data/" + ic + ".txt";
    string KE_file = "KE_Data/Data/" + ic + ".txt";
    ofstream particleData(particle_file, ios::out | ios::trunc);
    ofstream KEData(KE_file, ios::out | ios::trunc);

    host_particles = new double [10 * N];  // 3 each for position, velocity, force and 1 for mass
    hipMalloc(&gpu_particles, 10 * N * sizeof(double));

    for (int i = 0; i < N; i++) {
        host_particles[10 * i]     = particles[i].r[0];
        host_particles[10 * i + 1] = particles[i].r[1];
        host_particles[10 * i + 2] = particles[i].r[2];     // Positions
        
        host_particles[10 * i + 3] = particles[i].get_v()[0];
        host_particles[10 * i + 4] = particles[i].get_v()[1];
        host_particles[10 * i + 5] = particles[i].get_v()[2];   // Velocities
        
        host_particles[10 * i + 6] = particles[i].F[0];
        host_particles[10 * i + 7] = particles[i].F[1];
        host_particles[10 * i + 8] = particles[i].F[2];     // Forces
        
        host_particles[10 * i + 9] = (particles[i].type == 0) ? 1.0 : 10.0; // Mass
    }

    // Copy particle data from host to gpu
    hipMemcpy(gpu_particles, host_particles, 10 * N * sizeof(double), hipMemcpyHostToDevice);

    double* gpu_KE;
    hipMalloc(&gpu_KE, sizeof(double));
    hipMemset(gpu_KE, 0, sizeof(double));

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    // Compute initial KE
    computeKineticEnergy<<<blocks, threads>>>(gpu_particles, gpu_KE, N);
    hipDeviceSynchronize();
    
    double host_KE;
    hipMemcpy(&host_KE, gpu_KE, sizeof(double), hipMemcpyDeviceToHost);

    // Scale velocities if temperature is set
    if (temp != -1) {
        double lambda = sqrt((temp * 1.5 * 0.8314459920816467 * double(N)) / host_KE);
        scaleVelocities<<<blocks, threads>>>(gpu_particles, lambda, N);
        hipDeviceSynchronize();

        computeKineticEnergy<<<blocks, threads>>>(gpu_particles, gpu_KE, N);
        hipDeviceSynchronize();

        hipMemcpy(&host_KE, gpu_KE, sizeof(double), hipMemcpyDeviceToHost);
    }

    KEData << " " << 0.0 << " " << host_KE << endl;

    if (!ic_random) {
        hipMemcpy(host_particles, gpu_particles, 10 * N * sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i < N; i++) {
            particleData << " " << 0.0
                        << " " << i + 1
                        << " " << host_particles[10 * i]
                        << " " << host_particles[10 * i + 1]
                        << " " << host_particles[10 * i + 2]
                        << " " << host_particles[10 * i + 3]
                        << " " << host_particles[10 * i + 4]
                        << " " << host_particles[10 * i + 5] << endl;
        }
    }

    for (double t = dt; t < T + dt; t += dt) {
        updatePositions<<<blocks, threads>>>(gpu_particles, dt, N);
        hipDeviceSynchronize();

        calculateF_i<<<blocks, threads>>>(gpu_particles, N);
        hipDeviceSynchronize();

        updateVelocities<<<blocks, threads>>>(gpu_particles, dt, N, Lx, Ly, Lz);
        hipDeviceSynchronize();

        // Compute KE
        hipMemset(gpu_KE, 0, sizeof(double));
        computeKineticEnergy<<<blocks, threads>>>(gpu_particles, gpu_KE, N);
        hipDeviceSynchronize();
        hipMemcpy(&host_KE, gpu_KE, sizeof(double), hipMemcpyDeviceToHost);

        // Scale velocities if temperature is set
        if (temp != -1) {
            double lambda = sqrt((temp * 1.5 * 0.8314459920816467 * double(N)) / host_KE);
            scaleVelocities<<<blocks, threads>>>(gpu_particles, lambda, N);
            hipDeviceSynchronize();

            computeKineticEnergy<<<blocks, threads>>>(gpu_particles, gpu_KE, N);
            hipDeviceSynchronize();

            hipMemcpy(&host_KE, gpu_KE, sizeof(double), hipMemcpyDeviceToHost);
        }

        // Write to files every 0.1s
        if (fmod(t, 0.1) < dt) {
            KEData << " " << round(t * 10) / 10 << " " << host_KE << endl;
        }

        if (!ic_random) {
            if (fmod(t, 0.1) < dt) {
                hipMemcpy(host_particles, gpu_particles, 10 * N * sizeof(double), hipMemcpyDeviceToHost);
                for (int i = 0; i < N; i++) {
                    particleData << " " << round(t * 10) / 10
                                << " " << i + 1
                                << " " << host_particles[10 * i]
                                << " " << host_particles[10 * i + 1]
                                << " " << host_particles[10 * i + 2]
                                << " " << host_particles[10 * i + 3]
                                << " " << host_particles[10 * i + 4]
                                << " " << host_particles[10 * i + 5] << endl;
                }
            }
        }
    }

    hipFree(gpu_KE);
    delete[] host_particles;
    hipFree(gpu_particles);
    particleData.close();
    KEData.close();
}
